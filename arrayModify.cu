
#include <hip/hip_runtime.h>
#include  <stdio.h>

__global__ void modifyArray (int *modArray) {
    
    int i = threadIdx.x;
    modArray[i] = modArray[i] + 100;
}

__host__ int main (void) {
    
    int lenArray = 10;
    int *modArray, *gpu_modArray;
    size_t sizeArray;
    
    sizeArray = lenArray * sizeof(int);
    modArray  = (int*) malloc( sizeArray );
    hipMalloc( &gpu_modArray, sizeArray );
    
    printf("original values\n");
    for ( int i = 0; i < lenArray; i++ ) {
        modArray[i] = i + 1;
        printf("%d ", modArray[i]);
    }
    
    hipMemcpy( gpu_modArray, modArray, 
        sizeArray, hipMemcpyHostToDevice );
    
    modifyArray <<< 1, lenArray >>> (gpu_modArray);
    
    hipMemcpy( modArray, gpu_modArray, 
        sizeArray, hipMemcpyDeviceToHost );
    
    printf("\nfinal values\n");
    for ( int i = 0; i < lenArray; i++ )
        printf("%d ", modArray[i]);
    printf("\n");
    
    free( modArray );
    hipFree( gpu_modArray );
    
    return 0;
    
}
